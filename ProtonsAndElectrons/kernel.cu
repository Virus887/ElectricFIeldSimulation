#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/sequence.h>

#include "stdio.h"
#include <cmath>
#include <hip/hip_runtime.h>

#include "kernel.h"

#define TX 16
#define TY 16
#define GRIDDIM 70


#define BIG_DISPLAY_COEF 0.01f
#define DIST_COEF 0.01f

#define Q 0.5f



__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }


__global__
void displayKernel(uchar4* d_out, float2* fieldPowers, int w, int h,
    float2* pos_prot, float2* pos_elec, float2* big_pos_prot, float2* big_pos_elec,
    int howManyBigProt, int howManyBigElec,
    unsigned int* proton_begins, unsigned int* proton_ends, unsigned int* proton_indexes,
    unsigned int* electron_begins, unsigned int* electron_ends, unsigned int* electron_indexes, float BIG_PARTICLE_POWER) {

    //check if the pixel is the center of particle
    bool particle_flag = false;
    bool big_particle_flag = false;

    //get appropiate index
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ((i >= w) || (j >= h)) return; 
    const int ii = i + j * w; 

    //find current bucket index
    unsigned int x = (i / (float)w) * GRIDDIM;
    unsigned int y = (j / (float)h) * GRIDDIM;
    if (y >= h) y = GRIDDIM - 1;
    if (x >= w) x = GRIDDIM - 1;
    int bucket = y * GRIDDIM + x;
    if (bucket >= GRIDDIM * GRIDDIM) bucket = GRIDDIM * GRIDDIM - 1;


    bool b2 = (bucket / GRIDDIM != GRIDDIM - 1);           
    bool b4 = (bucket % GRIDDIM != 0);                    
    bool b6 = (bucket % GRIDDIM != GRIDDIM - 1);           
    bool b8 = (bucket / GRIDDIM != 0);                     
    bool b1 = (b2 && b4);
    bool b3 = (b6 && b2);
    bool b7 = (b8 && b4);
    bool b9 = (b8 && b6);

#pragma region collect_data
    //collect neighbour particles data
    float fieldPower = 0;
    for (int p = 0; p < howManyBigProt; p++)
    {
        float distx = big_pos_prot[p].x - i - 0.5f;
        float disty = big_pos_prot[p].y - j - 0.5f;
        float dist = sqrtf(distx * distx + disty * disty);
        if (dist <= 3.6f)
        {
            big_particle_flag = true;
        }
        if (dist < 0.01)continue;
        float F = BIG_PARTICLE_POWER * Q / (dist * dist * DIST_COEF);
        fieldPowers[ii].x += F * distx / dist;
        fieldPowers[ii].y += F * disty / dist;

        fieldPower += F * BIG_DISPLAY_COEF;
    }
    for (int p = 0; p < howManyBigElec; p++)
    {
        float distx = big_pos_elec[p].x - i - 0.5f;
        float disty = big_pos_elec[p].y - j - 0.5f;
        float dist = sqrtf(distx * distx + disty * disty);
        if (dist <= 3.6f)
        {
            big_particle_flag = true;
        }
        if (dist < 0.01)continue;
        float F = BIG_PARTICLE_POWER * Q / (dist * dist * DIST_COEF);
        fieldPowers[ii].x -= F * distx / dist;
        fieldPowers[ii].y -= F * disty / dist;
        fieldPower -= F * BIG_DISPLAY_COEF;
    }

    for (int p = proton_begins[bucket]; p < proton_ends[bucket]; p++)
    {
        int pp = proton_indexes[p];
        float distx = pos_prot[pp].x - i - 0.5f;
        float disty = pos_prot[pp].y - j - 0.5f;
        float dist = sqrtf(distx * distx + disty * disty);
        if (dist <= 0.6f)
        {
            particle_flag = true;
        }
        if (dist < 0.01)continue;
        float F = Q / (dist * dist * DIST_COEF);
        fieldPowers[ii].x += (F * distx) / dist;
        fieldPowers[ii].y += (F * disty) / dist;
        fieldPower += F;
    }
    for (int p = electron_begins[bucket]; p < electron_ends[bucket]; p++)
    {
        int pp = electron_indexes[p];
        float distx = pos_elec[pp].x - i - 0.5f;
        float disty = pos_elec[pp].y - j - 0.5f;
        float dist = sqrtf(distx * distx + disty * disty);
        if (dist <= 0.6f)
        {
            particle_flag = true;
        }
        if (dist < 0.01)continue;
        float F = Q / (dist * dist * DIST_COEF);
        fieldPowers[ii].x -= (F * distx) / dist;
        fieldPowers[ii].y -= (F * disty) / dist;

        fieldPower -= F;
    }


    if (b7)
    {
        for (int p = proton_begins[bucket - GRIDDIM- 1]; p < proton_ends[bucket- GRIDDIM - 1]; p++)
        {
            int pp = proton_indexes[p];
            float distx = pos_prot[pp].x - i - 0.5f;
            float disty = pos_prot[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x += F * distx / dist;
            fieldPowers[ii].y += F * disty / dist;

            fieldPower += F;
        }

        for (int p = electron_begins[bucket - GRIDDIM - 1]; p < electron_ends[bucket - GRIDDIM - 1]; p++)
        {
            int pp = electron_indexes[p];
            float distx = pos_elec[pp].x - i - 0.5f;
            float disty = pos_elec[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x -= F * distx / dist;
            fieldPowers[ii].y -= F * disty / dist;

            fieldPower -= F;
        }
    }

    if (b8)
    {
        for (int p = proton_begins[bucket - GRIDDIM]; p < proton_ends[bucket - GRIDDIM]; p++)
        {
            int pp = proton_indexes[p];
            float distx = pos_prot[pp].x - i - 0.5f;
            float disty = pos_prot[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x += F * distx / dist;
            fieldPowers[ii].y += F * disty / dist;

            fieldPower += F;
        }

        for (int p = electron_begins[bucket - GRIDDIM]; p < electron_ends[bucket - GRIDDIM]; p++)
        {
            int pp = electron_indexes[p];
            float distx = pos_elec[pp].x - i - 0.5f;
            float disty = pos_elec[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x -= F * distx / dist;
            fieldPowers[ii].y -= F * disty / dist;

            fieldPower -= F;
        }
    }

    if (b9)
    {
        for (int p = proton_begins[bucket - GRIDDIM + 1]; p < proton_ends[bucket - GRIDDIM + 1]; p++)
        {
            int pp = proton_indexes[p];
            float distx = pos_prot[pp].x - i - 0.5f;
            float disty = pos_prot[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x += F * distx / dist;
            fieldPowers[ii].y += F * disty / dist;

            fieldPower += F;
        }

        for (int p = electron_begins[bucket - GRIDDIM + 1]; p < electron_ends[bucket - GRIDDIM + 1]; p++)
        {
            int pp = electron_indexes[p];
            float distx = pos_elec[pp].x - i - 0.5f;
            float disty = pos_elec[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x -= F * distx / dist;
            fieldPowers[ii].y -= F * disty / dist;

            fieldPower -= F;
        }
    }

    if (b4)
    {
        for (int p = proton_begins[bucket- 1]; p < proton_ends[bucket - 1]; p++)
        {
            int pp = proton_indexes[p];
            float distx = pos_prot[pp].x - i - 0.5f;
            float disty = pos_prot[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x += F * distx / dist;
            fieldPowers[ii].y += F * disty / dist;

            fieldPower += F;
        }

        for (int p = electron_begins[bucket - 1]; p < electron_ends[bucket - 1]; p++)
        {
            int pp = electron_indexes[p];
            float distx = pos_elec[pp].x - i - 0.5f;
            float disty = pos_elec[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x -= F * distx / dist;
            fieldPowers[ii].y -= F * disty / dist;

            fieldPower -= F;
        }
    }

    if (b6)
    {
        for (int p = proton_begins[bucket + 1]; p < proton_ends[bucket + 1]; p++)
        {
            int pp = proton_indexes[p];
            float distx = pos_prot[pp].x - i - 0.5f;
            float disty = pos_prot[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x += F * distx / dist;
            fieldPowers[ii].y += F * disty / dist;

            fieldPower += F;
        }

        for (int p = electron_begins[bucket + 1]; p < electron_ends[bucket + 1]; p++)
        {
            int pp = electron_indexes[p];
            float distx = pos_elec[pp].x - i - 0.5f;
            float disty = pos_elec[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x -= F * distx / dist;
            fieldPowers[ii].y -= F * disty / dist;

            fieldPower -= F;
        }
    }

    if (b1)
    {
        for (int p = proton_begins[bucket + GRIDDIM - 1]; p < proton_ends[bucket + GRIDDIM - 1]; p++)
        {
            int pp = proton_indexes[p];
            float distx = pos_prot[pp].x - i - 0.5f;
            float disty = pos_prot[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x += F * distx / dist;
            fieldPowers[ii].y += F * disty / dist;

            fieldPower += F;
        }

        for (int p = electron_begins[bucket + GRIDDIM - 1]; p < electron_ends[bucket + GRIDDIM - 1]; p++)
        {
            int pp = electron_indexes[p];
            float distx = pos_elec[pp].x - i - 0.5f;
            float disty = pos_elec[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x -= F * distx / dist;
            fieldPowers[ii].y -= F * disty / dist;

            fieldPower -= F;
        }
    }

    if (b2)
    {
        for (int p = proton_begins[bucket + GRIDDIM]; p < proton_ends[bucket + GRIDDIM]; p++)
        {
            int pp = proton_indexes[p];
            float distx = pos_prot[pp].x - i - 0.5f;
            float disty = pos_prot[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x += F * distx / dist;
            fieldPowers[ii].y += F * disty / dist;

            fieldPower += F;
        }

        for (int p = electron_begins[bucket + GRIDDIM]; p < electron_ends[bucket + GRIDDIM]; p++)
        {
            int pp = electron_indexes[p];
            float distx = pos_elec[pp].x - i - 0.5f;
            float disty = pos_elec[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x -= F * distx / dist;
            fieldPowers[ii].y -= F * disty / dist;

            fieldPower -= F;
        }
    }

    if (b3)
    {
        for (int p = proton_begins[bucket + GRIDDIM + 1]; p < proton_ends[bucket + GRIDDIM + 1]; p++)
        {
            int pp = proton_indexes[p];
            float distx = pos_prot[pp].x - i - 0.5f;
            float disty = pos_prot[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x += F * distx / dist;
            fieldPowers[ii].y += F * disty / dist;

            fieldPower += F;
        }

        for (int p = electron_begins[bucket + GRIDDIM + 1]; p < electron_ends[bucket + GRIDDIM + 1]; p++)
        {
            int pp = electron_indexes[p];
            float distx = pos_elec[pp].x - i - 0.5f;
            float disty = pos_elec[pp].y - j - 0.5f;
            float dist = sqrtf(distx * distx + disty * disty);
            if (dist < 0.01)continue;

            float F = Q / (dist * dist * DIST_COEF);
            fieldPowers[ii].x -= F * distx / dist;
            fieldPowers[ii].y -= F * disty / dist;

            fieldPower -= F;
        }
    }
#pragma endregion


    d_out[ii].y = 0;
    d_out[ii].w = 255;

    float disp_coef = 5.0f;
    if (fieldPower > 0)
    {
        const unsigned char intensity = clip(fieldPower * disp_coef);
        d_out[ii].x = intensity;//red
        d_out[ii].z = 0; //blue
    }
    else
    {
        const unsigned char intensity = clip(-fieldPower * disp_coef);
        d_out[ii].x = 0; //red
        d_out[ii].z = intensity; //blue
    }

    //put black dot inside small particle
    if (particle_flag)
    {
        d_out[ii].x = 0;
        d_out[ii].z = 0;
    }

    //put white dot inside big particle
    else if (big_particle_flag)
    {
        d_out[ii].y = 255;
        d_out[ii].w = 255;
        d_out[ii].x = 255;
        d_out[ii].z = 255;
    }
}


__global__
void movementKernel(int w, int h, float2* pos_prot, float2* pos_elec, float2* v_prot, float2* v_elec, float2* fieldPowers,
    int howManyProt, int howManyElec, int howManyBigProt, int howManyBigElec, float dt) {
    float ax = 0, ay = 0;
    
    int ii = blockIdx.x * blockDim.x + threadIdx.x; // 1D indexing
    if (ii >= howManyProt + howManyElec) return; // Check if within image bounds

    if (ii < howManyProt) // I am proton
    {   
        unsigned int x = floor(pos_prot[ii].x);
        unsigned int y = floor(pos_prot[ii].y);
        unsigned int pixel = y * w + x;


        ax -= Q * fieldPowers[pixel].x ;
        ay -= Q * fieldPowers[pixel].y ;

        v_prot[ii].x += ax * dt;
        v_prot[ii].y += ay * dt;


        pos_prot[ii].x += v_prot[ii].x * dt;
        pos_prot[ii].y += v_prot[ii].y * dt;

        //BOUNCE
        if (pos_prot[ii].x >= w)
        {
            v_prot[ii].x = -v_prot[ii].x / 2.0f;
            pos_prot[ii].x = w - 1.0f;
        }
        if (pos_prot[ii].y >= h)
        {
            v_prot[ii].y = -v_prot[ii].y / 2.0f;
            pos_prot[ii].y = h - 1.0f;
        }
        if (pos_prot[ii].x <= 0)
        {
            v_prot[ii].x = -v_prot[ii].x / 2.0f;
            pos_prot[ii].x = 1.0f;
        }
        if (pos_prot[ii].y <= 0)
        {
            v_prot[ii].y = -v_prot[ii].y / 2.0f;
            pos_prot[ii].y = 1.0f;
        }

    }
    else//I am electron
    {
        ii -= howManyProt;

        unsigned int x = floor(pos_elec[ii].x);
        unsigned int y = floor(pos_elec[ii].y);
        if (x >= w) x = w - 1;
        if (y >= h) y = h - 1;
        int pixel = y * w + x;

        ax += fieldPowers[pixel].x ;
        ay += fieldPowers[pixel].y ;


        v_elec[ii].x += ax * dt;
        v_elec[ii].y += ay * dt;

        pos_elec[ii].x += v_elec[ii].x * dt;
        pos_elec[ii].y += v_elec[ii].y * dt;

        //BOUNCE
        if (pos_elec[ii].x >= w)
        {
            v_elec[ii].x = -v_elec[ii].x / 2.0f;
            pos_elec[ii].x = w-1.0f;
        }
        if (pos_elec[ii].y >= h)
        {
            v_elec[ii].y = -v_elec[ii].y / 2.0f;
            pos_elec[ii].y = h-1.0f;
        }
        if (pos_elec[ii].x <= 0)
        {
            v_elec[ii].x = -v_elec[ii].x / 2.0f;
            pos_elec[ii].x = 1.0f;
        }
        if (pos_elec[ii].y <= 0)
        {
            v_elec[ii].y = -v_elec[ii].y / 2.0f;
            pos_elec[ii].y = 1.0f;
        }

    }
}


struct calculate_bucket_index
{
    float w, h;
    int gridDim;
    __host__ __device__
        calculate_bucket_index(float width, float height, int dim)
        :w(width), h(height), gridDim(dim) {}
    __host__ __device__
        unsigned int operator()(float2 p) const
    {
        // coordinates of the grid cell containing point p
        unsigned int x = ((p.x / w) * gridDim);
        unsigned int y = ((p.y / h) * gridDim);
        // return the bucket's linear index
        if (y * gridDim + x >= gridDim * gridDim) return gridDim * gridDim - 1;
        else return y * gridDim + x;
    }
};



thrust::device_vector<float2> d_p;// = particles->p_pos;
thrust::device_vector<float2> d_e;// = particles->e_pos;
thrust::device_vector<float2> d_big_p;// = particles->big_p_pos;
thrust::device_vector<float2> d_big_e;// = particles->big_e_pos;
thrust::device_vector<float2> d_pv;// = particles->p_v;
thrust::device_vector<float2> d_ev;// = particles->e_v;


float2* ptr_protons;//= thrust::raw_pointer_cast(&d_p[0]);
float2* ptr_electrons;//= thrust::raw_pointer_cast(&d_e[0]);
float2* ptr_big_protons;//= thrust::raw_pointer_cast(&d_big_p[0]);
float2* ptr_big_electrons;// = thrust::raw_pointer_cast(&d_big_e[0]);
float2* ptr_protons_v;//= thrust::raw_pointer_cast(&d_pv[0]);
float2* ptr_electrons_v;//= thrust::raw_pointer_cast(&d_ev[0]);


void prepareParticles(Particles* particles)
{
    d_p = particles->p_pos;
    d_e = particles->e_pos;
    d_big_p = particles->big_p_pos;
    d_big_e = particles->big_e_pos;
    d_pv = particles->p_v;
    d_ev = particles->e_v;

    ptr_protons = thrust::raw_pointer_cast(&d_p[0]);
    ptr_electrons = thrust::raw_pointer_cast(&d_e[0]);
    ptr_big_protons = thrust::raw_pointer_cast(&d_big_p[0]);
    ptr_big_electrons = thrust::raw_pointer_cast(&d_big_e[0]);
    ptr_protons_v = thrust::raw_pointer_cast(&d_pv[0]);
    ptr_electrons_v = thrust::raw_pointer_cast(&d_ev[0]);

}


void kernelLauncher(uchar4* d_out, Particles* particles, float DT, float BIG_PARTICLE_POWER) {
    const dim3 blockSize(TX, TY);
    const dim3 gridSize = dim3((particles->w + TX - 1) / TX, (particles->h + TY - 1) / TY);
    hipError_t err;

#pragma region thrust

    //PROTONS
    //bucket sort using thrust library
    //give each proton its bucket index 
    thrust::device_vector<unsigned int> proton_sorted_indexes = particles->p_index;

    thrust::device_vector<unsigned int> proton_bucket_indices(particles->howManyProtons);
    thrust::transform(d_p.begin(), d_p.end(), proton_bucket_indices.begin(), calculate_bucket_index(particles->w, particles->h, GRIDDIM));

    //sort protons indexes by bucket
    thrust::sort_by_key(proton_bucket_indices.begin(), proton_bucket_indices.end(), proton_sorted_indexes.begin());

    //return each bucket begin and end
    thrust::device_vector<unsigned int> proton_bucket_start(GRIDDIM * GRIDDIM);
    thrust::device_vector<unsigned int> proton_bucket_end(GRIDDIM * GRIDDIM);

    thrust::counting_iterator<unsigned int> proton_search_begin(0);
    thrust::lower_bound(proton_bucket_indices.begin(), proton_bucket_indices.end(), proton_search_begin, proton_search_begin + GRIDDIM * GRIDDIM, proton_bucket_start.begin());  //returns iterator on first proton with bucket index >= i
    thrust::upper_bound(proton_bucket_indices.begin(), proton_bucket_indices.end(), proton_search_begin, proton_search_begin + GRIDDIM * GRIDDIM, proton_bucket_end.begin());  //returns iterator on first proton with bucket index > i 
    
    
    unsigned int* ptr_proton_starts = thrust::raw_pointer_cast(&proton_bucket_start[0]);
    unsigned int* ptr_proton_ends = thrust::raw_pointer_cast(&proton_bucket_end[0]);
    unsigned int* ptr_proton_sorted_indexes = thrust::raw_pointer_cast(&proton_sorted_indexes[0]);


    //ELECTRONS
    //repeat for electrons
    thrust::device_vector<unsigned int> electron_sorted_indexes = particles->e_index;

    thrust::device_vector<unsigned int> electron_bucket_indices(particles->howManyElectrons);
    thrust::transform(d_e.begin(), d_e.end(), electron_bucket_indices.begin(), calculate_bucket_index(particles->w, particles->h, GRIDDIM));

    //sort boid indexes by bucket
    thrust::sort_by_key(electron_bucket_indices.begin(), electron_bucket_indices.end(), electron_sorted_indexes.begin());

    //return each bucket begin and end
    thrust::device_vector<unsigned int> electron_bucket_start(GRIDDIM * GRIDDIM);
    thrust::device_vector<unsigned int> electron_bucket_end(GRIDDIM * GRIDDIM);

    thrust::counting_iterator<unsigned int> electron_search_begin(0);
    thrust::lower_bound(electron_bucket_indices.begin(), electron_bucket_indices.end(), electron_search_begin, electron_search_begin + GRIDDIM * GRIDDIM, electron_bucket_start.begin());  //returns iterator on first electron with bucket index >= i
    thrust::upper_bound(electron_bucket_indices.begin(), electron_bucket_indices.end(), electron_search_begin, electron_search_begin + GRIDDIM * GRIDDIM, electron_bucket_end.begin());  //returns iterator on first electron with bucket index > i 
    
    
    unsigned int* ptr_electron_starts = thrust::raw_pointer_cast(&electron_bucket_start[0]);
    unsigned int* ptr_electron_ends = thrust::raw_pointer_cast(&electron_bucket_end[0]);
    unsigned int* ptr_electron_sorted_indexes = thrust::raw_pointer_cast(&electron_sorted_indexes[0]);


    thrust::device_vector<float2> field_power_vec = thrust::device_vector<float2>(particles->w * particles->h);
    float2* ptr_field_powers = thrust::raw_pointer_cast(&field_power_vec[0]);

#pragma endregion

    displayKernel <<<gridSize, blockSize >> > 
        (d_out,ptr_field_powers ,particles->w, particles->h, 
         ptr_protons, ptr_electrons, ptr_big_protons, ptr_big_electrons,
         particles->howManyBigProtons, particles->howManyBigElectrons,
         ptr_proton_starts, ptr_proton_ends, ptr_proton_sorted_indexes, ptr_electron_starts, ptr_electron_ends, ptr_electron_sorted_indexes, BIG_PARTICLE_POWER);


    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed Display (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (particles->howManyProtons + particles->howManyElectrons + threadsPerBlock - 1) / threadsPerBlock;
    movementKernel << <blocksPerGrid, threadsPerBlock >> >
            (particles->w, particles->h, ptr_protons, ptr_electrons, ptr_protons_v, ptr_electrons_v, ptr_field_powers,
                particles->howManyProtons, particles->howManyElectrons, particles->howManyBigProtons, particles->howManyBigElectrons, DT);


    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed Movement (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    particles->p_pos = d_p;
    particles->e_pos = d_e;
    particles->p_v = d_pv;
    particles->e_v = d_ev;
}

